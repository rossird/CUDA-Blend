#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */

#include "utils.h"
#include <thrust/host_vector.h>
#include "reference_calc.cpp"
#include <cstdio>


/*** DEBUG FUNCTIONS ***/
void printDeviceArray(bool* d_array, int numRows, int numCols)
{

  int size = numRows * numCols;
  bool* h_array = new bool[size];
  hipMemcpy(h_array, d_array, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < numRows; i++){
    for(int j = 0; j < numCols; j++){ 
      printf("%i", h_array[i * numCols + j]);
    }
    printf("\n");
  }

  delete h_array;
  h_array = NULL;

}
/********************/


/** Calculate the mask pixels 
 *
 *  Pixels that have a value of 255 for R, G, and B are mask pixels.
 *  
 *  Outputs an array of 0/1s, where 1s means that pixel should be copied.
 **/
 __global__
void maskKernel(uchar4* d_source, bool* d_mask, size_t numRows, size_t numCols)
{
  //One thread per pixel
  int g_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(g_id > numRows * numCols - 1)
    return;
  uchar4 pix = d_source[g_id];
  d_mask[g_id] = !(pix.x == 255 && pix.y == 255 && pix.z == 255);
}

__global__
void borderPredicateKernel(bool* d_mask,
                           bool* d_border,
                           bool* d_interior,
                           size_t numRows,
                           size_t numCols)
{

  int g_id = threadIdx.x + blockIdx.x * blockDim.x ;
  if(g_id > numRows * numCols - 1)
    return;

  if(d_mask[g_id]) {

    int curCol = threadIdx.x;
    int curRow = blockIdx.x; //Assuming kernel is called with 1 block per row.

    //Calculate neighbors
    int maskedNeighbors = 0;
    //Up
    if(curRow > 0) {
      maskedNeighbors += d_mask[curCol + (curRow - 1) * numCols];
    }
    //Down
    if(curRow < numRows - 1) {
      maskedNeighbors += d_mask[curCol + (curRow + 1) * numCols];
    }
    //Left
    if(curCol > 0) {
      maskedNeighbors += d_mask[curCol - 1 + curRow * numCols];
    }
    //Right
    if(curCol < numCols - 1) {
      maskedNeighbors += d_mask[curCol + 1 + curRow * numCols];
    }

    //Interior if all four neighbors are also in mask
    //Border if in mask, but at least one neighbor is not in mask
    if(maskedNeighbors >= 4) {
      d_interior[g_id] = true;
    } else {
      d_border[g_id] = true;
    }
  }

}

//Taken from problem set 5
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      float* const redChannel,
                      float* const greenChannel,
                      float* const blueChannel)
{
  int g_id = threadIdx.x + blockIdx.x * blockDim.x;

  uchar4 rgba = inputImageRGBA[g_id];
  redChannel[g_id] = (float)rgba.x;
  greenChannel[g_id] = (float)rgba.y;
  blueChannel[g_id] = (float)rgba.z;
}

//Taken from homework 5
__global__
void recombineChannels(const float* const redChannel,
                       const float* const greenChannel,
                       const float* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  int g_id = threadIdx.x + blockIdx.x * blockDim.x;
  if (g_id > numRows * numCols - 1) 
    return;
  outputImageRGBA[g_id].x = (char)redChannel[g_id];
  outputImageRGBA[g_id].y = (char)greenChannel[g_id];
  outputImageRGBA[g_id].z = (char)blueChannel[g_id];

}

__global__
void  jacobiKernel(float* d_in,
                   float* d_out,
                   float* d_sourceChannel,
                   float* d_destChannel,
                   bool* d_border,
                   bool* d_interior,
                   size_t numRows,
                   size_t numCols)
{
  
  int g_id = threadIdx.x + blockIdx.x * blockDim.x;
  if(g_id > numRows * numCols - 1) 
    return;

  if (d_interior[g_id]){
    /*
       1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
          Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

          Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

      2) Calculate the new pixel value:
          float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
          ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]
      */
    float sum1 = 0;
    float sum2 = 0;

    int curCol = threadIdx.x;
    int curRow = blockIdx.x; //Assuming kernel is called with 1 block per row.

    //Lets save this so we only have to access global once instead of (upto) 4 times.
    float sourceValue = d_sourceChannel[g_id];

    //Up
    if(curRow > 0) {
      int neighbor_idx = curCol + (curRow - 1) * numCols;
      if (d_interior[neighbor_idx]) {
        sum1 += d_in[neighbor_idx];
      } else if (d_border[neighbor_idx]) {
        sum1 += d_destChannel[neighbor_idx];
      }      
      sum2 += sourceValue - d_sourceChannel[neighbor_idx];
    }
    //Down
    if(curRow < numRows - 1) {
      int neighbor_idx = curCol + (curRow + 1) * numCols; 
      if (d_interior[neighbor_idx]) {
        sum1 += d_in[neighbor_idx];
      } else if (d_border[neighbor_idx]) {
        sum1 += d_destChannel[neighbor_idx];
      }      
      sum2 += sourceValue - d_sourceChannel[neighbor_idx];
    }
    //Left
    if(curCol > 0) {
      int neighbor_idx = curCol - 1 + curRow * numCols; 
      if (d_interior[neighbor_idx]) {
        sum1 += d_in[neighbor_idx];
      } else if (d_border[neighbor_idx]) {
        sum1 += d_destChannel[neighbor_idx];
      }      
      sum2 += sourceValue - d_sourceChannel[neighbor_idx];
    }
    //Right
    if(curCol < numCols - 1) {
      int neighbor_idx = curCol + 1 + curRow * numCols; 
      if (d_interior[neighbor_idx]) {
        sum1 += d_in[neighbor_idx];
      } else if (d_border[neighbor_idx]) {
        sum1 += d_destChannel[neighbor_idx];
      }      
      sum2 += sourceValue - d_sourceChannel[neighbor_idx];
    }

    float newVal= (sum1 + sum2) / 4.f;
    d_out[g_id] = min(255.f, max(0.f, newVal)); //clamp to [0, 255]

  } else {
    //Not an interior pixel, so just set output to input
    d_out[g_id] = d_destChannel[g_id];
  }
}




void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

    //Allocate device pointers for each image
    uchar4* d_source;
    uchar4* d_dest;
    uchar4* d_blended;  //This is the output image

    int imgSize = numRowsSource * numColsSource * sizeof(uchar4);

    hipMalloc(&d_source, imgSize);
    hipMalloc(&d_dest, imgSize);
    hipMalloc(&d_blended, imgSize);

    //Move images to device memory
    hipMemcpy(d_source, h_sourceImg, imgSize, hipMemcpyHostToDevice);
    hipMemcpy(d_dest, h_destImg, imgSize, hipMemcpyHostToDevice);

    //Mask
    bool* d_mask;
    bool* d_borderPred;
    bool* d_interiorPred;
    
    int maskSize = numRowsSource * numColsSource * sizeof(bool);
    
    hipMalloc(&d_mask, maskSize);
    hipMalloc(&d_borderPred, maskSize);
    hipMalloc(&d_interiorPred, maskSize);

    hipMemset(d_borderPred, 0, maskSize);
    hipMemset(d_interiorPred, 0, maskSize);
    

  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.
  */

    maskKernel<<<numRowsSource, numColsSource>>>(d_source,
                                                 d_mask,
                                                 numRowsSource,
                                                 numColsSource);
    //This part looks pretty good (10:15am 11/12/2016)
    //printDeviceArray(d_mask, numRowsSource, numColsSource);

  /*
     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.
  */

    borderPredicateKernel<<<numRowsSource, numColsSource>>>(d_mask,
                                                            d_borderPred,
                                                            d_interiorPred,
                                                            numRowsSource,
                                                            numColsSource);
    //printDeviceArray(d_borderPred, numRowsSource, numColsSource);
    //printDeviceArray(d_interiorPred, numRowsSource, numColsSource);
    //This part looks good (10:56am 11/12/2016)

  /*

     3) Separate out the incoming image into three separate channels

  */

    float* d_sourceRed;
    float* d_sourceGreen;
    float* d_sourceBlue;

    float* d_destRed;
    float* d_destGreen;
    float* d_destBlue;

    int channelSize = numRowsSource * numColsSource * sizeof(float);

    checkCudaErrors(hipMalloc(&d_sourceRed, channelSize));
    checkCudaErrors(hipMalloc(&d_sourceGreen, channelSize));
    checkCudaErrors(hipMalloc(&d_sourceBlue, channelSize));

    checkCudaErrors(hipMalloc(&d_destRed, channelSize));
    checkCudaErrors(hipMalloc(&d_destGreen, channelSize));
    checkCudaErrors(hipMalloc(&d_destBlue, channelSize));

    separateChannels<<<numRowsSource, numColsSource>>>(d_source,
                                                       numRowsSource,
                                                       numColsSource,
                                                       d_sourceRed,
                                                       d_sourceGreen,
                                                       d_sourceBlue);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    separateChannels<<<numRowsSource, numColsSource>>>(d_dest,
                                                       numRowsSource,
                                                       numColsSource,
                                                       d_destRed,
                                                       d_destGreen,
                                                       d_destBlue);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  /*
     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.
  */
    float* d_red1;
    float* d_red2;
    float* d_green1;
    float* d_green2;
    float* d_blue1;
    float* d_blue2;

    checkCudaErrors(hipMalloc(&d_red1, channelSize));
    checkCudaErrors(hipMalloc(&d_red2, channelSize));
    checkCudaErrors(hipMalloc(&d_green1, channelSize));
    checkCudaErrors(hipMalloc(&d_green2, channelSize));
    checkCudaErrors(hipMalloc(&d_blue1, channelSize));
    checkCudaErrors(hipMalloc(&d_blue2, channelSize));

    checkCudaErrors(hipMemcpy(d_red1, d_sourceRed, channelSize, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_green1, d_sourceGreen, channelSize, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_blue1, d_sourceBlue, channelSize, hipMemcpyDeviceToDevice));

  /*
     5) For each color channel perform the Jacobi iteration described 
        above 800 times.
  */

    for(int i = 0; i < 800; i++) {

      jacobiKernel<<<numRowsSource, numColsSource>>>(d_red1,
                                                     d_red2,
                                                     d_sourceRed,
                                                     d_destRed,
                                                     d_borderPred,
                                                     d_interiorPred,
                                                     numRowsSource,
                                                     numColsSource);

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      jacobiKernel<<<numRowsSource, numColsSource>>>(d_green1,
                                                     d_green2,
                                                     d_sourceGreen,
                                                     d_destGreen,
                                                     d_borderPred,
                                                     d_interiorPred,
                                                     numRowsSource,
                                                     numColsSource);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      jacobiKernel<<<numRowsSource, numColsSource>>>(d_blue1,
                                                     d_blue2,
                                                     d_sourceBlue,
                                                     d_destBlue,
                                                     d_borderPred,
                                                     d_interiorPred,
                                                     numRowsSource,
                                                     numColsSource);

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      //Swap buffers 1 and 2 for each color
      float* temp = d_red1;
      d_red1 = d_red2;
      d_red2 = temp;

      temp = d_green1;
      d_green1 = d_green2;
      d_green2 = temp;

      temp = d_blue1;
      d_blue1 = d_blue2;
      d_blue2 = temp;
    }

  /*

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.
  */

    recombineChannels<<<numRowsSource, numColsSource>>>(d_red1,
                                                        d_green1,
                                                        d_blue1,
                                                        d_blended,
                                                        numRowsSource,
                                                        numColsSource);

    hipMemcpy(h_blendedImg, d_blended, imgSize, hipMemcpyDeviceToHost);

  /*
      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */

    hipFree(d_source);
    hipFree(d_dest);
    hipFree(d_blended);

    hipFree(d_mask);
    hipFree(d_borderPred);
    hipFree(d_interiorPred);

    hipFree(d_sourceRed);
    hipFree(d_sourceGreen);
    hipFree(d_sourceBlue);
    hipFree(d_destRed);
    hipFree(d_destGreen);
    hipFree(d_destBlue);

    hipFree(d_red1);
    hipFree(d_red2);
    hipFree(d_green1);
    hipFree(d_green2);
    hipFree(d_blue1);
    hipFree(d_blue2);

  /* The reference calculation is provided below, feel free to use it
     for debugging purposes. 
   */

  /*
    uchar4* h_reference = new uchar4[srcSize];
    reference_calc(h_sourceImg, numRowsSource, numColsSource,
                   h_destImg, h_reference);

    checkResultsEps((unsigned char *)h_reference, (unsigned char *)h_blendedImg, 4 * srcSize, 2, .01);
    delete[] h_reference; */
}
